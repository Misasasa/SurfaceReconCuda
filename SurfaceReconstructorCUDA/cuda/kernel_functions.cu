#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include "kernel_functions.h"
#include "SPHHelper.hpp"

#include "MarchingCube.h"

__constant__ int device_indexMap[1024];
__constant__ SPHHelper device_sphhelper;
__constant__ cint3 device_neighborCells[27];

__inline__ __device__ int GetCellHash(cint3 pCoord, int resolution) {
	if (pCoord.x<0 || pCoord.x>=resolution ||
		pCoord.y<0 || pCoord.y>=resolution ||
		pCoord.z<0 || pCoord.z>=resolution)
		return INVALID_CELL;

	int mappedIndex = device_indexMap[pCoord.x] |
		device_indexMap[pCoord.y]<<1 |
		device_indexMap[pCoord.z]<<2;
	return mappedIndex;
}

__global__ void ComputeParticleHash(ZIndexGridCUDA zgrid) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= zgrid.numParticles)
		return;
	auto & pos = zgrid.particles[i].pos;
	auto coord = GetCoordinate(pos, zgrid.xmin, zgrid.cellWidth);
	auto hash = GetCellHash(coord, zgrid.resolution);
	zgrid.particleHashes[i] = hash;
	zgrid.particleIndices[i] = i;
}


__global__ void ReorderDataAndFindCellStart(ZIndexGridCUDA zgrid) {

	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	extern __shared__ uint sharedHash[];
	uint hash;

	if (index < zgrid.numParticles)
	{
		hash = zgrid.particleHashes[index];

		// Load hash data into shared memory so that we can look
		// at neighboring particle's hash value without loading
		// two hash values per thread
		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0)
		{
			// first thread in block must load neighbor particle hash
			sharedHash[0] = zgrid.particleHashes[index - 1];
		}
	}

	__syncthreads();


	if (index < zgrid.numParticles)
	{
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell,
		// so store the index of this particle in the cell.
		// As it isn't the first particle, it must also be the cell end of
		// the previous particle's cell

		if (index == 0 || hash != sharedHash[threadIdx.x])
		{
			if (hash!=INVALID_CELL)
				zgrid.startIndices[hash] = index;

			if (index > 0)
				zgrid.endIndices[sharedHash[threadIdx.x]] = index;
		}
		if (index == zgrid.numParticles - 1)
		{
			if (hash != INVALID_CELL)
				zgrid.endIndices[hash] = index + 1;
		}


		// Now use the sorted index to reorder the pos and vel data
		uint sortedIndex = zgrid.particleIndices[index];
		auto p = zgrid.particles[sortedIndex];
		zgrid.reorderBuffer[index] = p;
	}
}



__global__ void ComputeColorField(
	ZIndexGridCUDA zgrid,
	float spacing,
	float infectRadius,
	float normThres,
	int neighborThres,
	int* surfaceParticleMark
) {
	uint i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if(i>=zgrid.numParticles)
		return;
	
	auto& xi = zgrid.particles[i].pos;
	auto coord = GetCoordinate(xi, zgrid.xmin, zgrid.cellWidth);
	auto hash = zgrid.particleHashes[i];

	int numNeighbors=0;
	cfloat3 normal(0,0,0);
	float vol = spacing*spacing*spacing;

	for (int xx=-1; xx<=1; xx++)
		for (int yy=-1; yy<=1; yy++)
			for (int zz=-1; zz<=1; zz++) {
				auto coord1 = coord + cint3(xx, yy, zz);
				auto hash1 = GetCellHash(coord1, zgrid.resolution);
				if(hash1 == INVALID_CELL)
					continue;

				int startIndex = zgrid.startIndices[hash1];
				if(startIndex == CELL_EMPTY)
					continue;
				int endIndex = zgrid.endIndices[hash1];
				for (int j=startIndex; j<endIndex; j++) {
					if(j==i)
						continue;
					auto& xj = zgrid.particles[j].pos;
					auto xij = xi - xj;
					auto d = xij.Norm();
					if(d>=infectRadius)
						continue;
					auto nablaw = device_sphhelper.CubicGradient(xij);
					normal += nablaw;
					numNeighbors++;
				}
			}
	normal = normal*vol;
	auto nnorm = normal.Norm();
	if(nnorm > normThres || numNeighbors<neighborThres)
		surfaceParticleMark[i] = 1;
	else
		surfaceParticleMark[i] = 0;
}


__global__ void  ComputeScalarValues(
	ZIndexGridCUDA zgrid,
	SurfaceGridCUDA sgrid,
	float particleSpacing,
	float infectRadius
) {
	uint i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if(i >= sgrid.numSurfaceVertices)
		return;

	auto& vertex = sgrid.device_surfaceVertices[i];
	auto gridIndex = vertex.gridIndex;
	auto coord = GetCoordinate(gridIndex, sgrid.vertexResolution);
	auto xi = GetPosition(coord, sgrid.xmin, sgrid.cellWidth);

	auto zcoord = GetCoordinate(xi, zgrid.xmin, zgrid.cellWidth);
	auto zhash = GetCellHash(zcoord, zgrid.resolution);
	if(zhash == INVALID_CELL)
		return;

	float pVol = particleSpacing * particleSpacing * particleSpacing;
	cfloat3 xAverage(0,0,0);
	cmat3 xAverageGradient;
	float sumW = 0;
	cfloat3 sumNablaW(0,0,0);

	for (int i=0; i<27; i++) {
		auto coord1 = zcoord + device_neighborCells[i];
		auto hash1 = GetCellHash(coord1, zgrid.resolution);
		if(hash1 == INVALID_CELL)
			continue;
		
		int startIndex = zgrid.startIndices[hash1];
		if (startIndex == CELL_EMPTY)
			continue;
		int endIndex = zgrid.endIndices[hash1];

		// for each neighboring particle
		for (int j=startIndex; j<endIndex; j++) {

			auto& xj = zgrid.particles[j].pos;
			auto xij = xi - xj;
			auto d = xij.Norm();
			if (d >= infectRadius)
				continue;
			float w_ij = device_sphhelper.Cubic(d);
			cfloat3 nablaW = device_sphhelper.CubicGradient(xij);
			xAverage += xj * w_ij;
			sumW += w_ij;
			sumNablaW += nablaW;

			xAverageGradient.Add(TensorProduct(xj, nablaW));
		}
	}

	float scalarValue;
	if (abs(sumW)>EPSILON) {
		xAverage /= sumW;
		scalarValue = (xi - xAverage).Norm() - particleSpacing;
	}
	else
		scalarValue = OUTSIDE;
	vertex.value = scalarValue;
}