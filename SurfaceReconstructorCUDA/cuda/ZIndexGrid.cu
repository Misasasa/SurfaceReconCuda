#include "ZIndexGrid.cuh"
#include "cuda_common.cuh"
#include "hip/hip_runtime_api.h"


extern __constant__ int device_indexMap[1024];
extern int indexMap[1024];

void ZIndexGridCUDA::AllocateDeviceBuffer() {
	hipMalloc(&startIndices, sizeof(uint)*numCells);
	hipMalloc(&endIndices, sizeof(uint)*numCells);
	hipMemcpyToSymbol(HIP_SYMBOL(device_indexMap), indexMap, sizeof(indexMap));
}

