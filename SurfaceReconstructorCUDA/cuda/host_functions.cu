

#include "kernel_functions.h"
#include "cuda_common.h"
#include "hip/hip_runtime_api.h"

void ReorderDataAndFindCellStart_Host(ZIndexGridCUDA& zgrid) {
	int numBlocks, numThreads;
	computeBlockSize(zgrid.numParticles, 256, numBlocks, numThreads);

	hipMemset(zgrid.startIndices, CELL_EMPTY, zgrid.numCells*sizeof(uint));
	int sharedMemSize = sizeof(int)*(numThreads+1);

	ReorderDataAndFindCellStart <<< numBlocks, numThreads, sharedMemSize>>>(zgrid);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed: reorder data");
	hipMemcpy(zgrid.particles, zgrid.reorderBuffer, sizeof(Particle)*zgrid.numParticles, hipMemcpyDeviceToDevice);
}



void ComputeParticleHash_Host(ZIndexGridCUDA& zgrid) {

	int numBlocks, numThreads;
	computeBlockSize(zgrid.numParticles, 256, numBlocks, numThreads);

	ComputeParticleHash <<<numBlocks, numThreads>>> (zgrid);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed:compute particle hash");
}

void ComputeColorField_Host(
	ZIndexGridCUDA& zgrid,
	float spacing,
	float infectRadius,
	float normThres,
	int neighborThres,
	int* surfaceParticleMark) {
	
	int numBlocks, numThreads;
	computeBlockSize(zgrid.numParticles, 256, numBlocks, numThreads);

	ComputeColorField<<<numBlocks, numThreads>>>(zgrid,
		spacing,
		infectRadius,
		normThres,
		neighborThres,
		surfaceParticleMark);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed:compute color field");
}

void ComputeScalarValues_Host(
	ZIndexGridCUDA& zgrid,
	SurfaceGridCUDA& sgrid,
	float particleSpacing,
	float infectRadius
) {
	int numBlocks, numThreads;
	computeBlockSize(sgrid.numSurfaceVertices, 256, numBlocks, numThreads);

	ComputeScalarValues<<<numBlocks, numThreads>>>(
		zgrid,
		sgrid,
		particleSpacing,
		infectRadius);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed:compute scalar values");
}